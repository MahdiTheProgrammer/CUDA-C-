#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixmultiplication(int *a, int *b, int *c, int x, int y){
 

}

int main(){

	// Initializing two matrices with shape of x,m and m,y
	int x = 32;
	int m = 64;
	int y = 72;

	int size_A = x*m*sizeof(int);
	int size_B = m*y*sizeof(int);
	int size_C = x*y*sizeof(int);

	int *h_A = new int[x*m];
	int *h_B = new int[m*y];
	int *h_C = new int[x*y];

	for (int f1=0; f1<x; f1++){
		for (int f2=0; f2<m;f2++){
			h_A[f1*x + f2] = f1;
		}
	}


	for (int f1=0; f1<m; f1++){
		for (int f2=0; f2<y;f2++){
			h_B[f1*m + f2] = f1;
		}
	}

	for (int val : h_A) {
		cout << val << " ";
	}

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, size_A);
	hipMalloc((void**)&d_B, size_B);
	hipMalloc((void**)&d_C, size_C);

	hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);

	dim3 blockDim(32,32);
	dim3 gridDim((y+31)/32,(x+31)/32);
	matrixmultiplication<<<blockDim,gridDim>>>(d_A,d_B,d_C,x,y);
}
//this is not ready yet...................soon
