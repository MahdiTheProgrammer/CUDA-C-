#include "hip/hip_runtime.h"
#include "layer.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void convolution(float* input,float*weights, float* bias, float* output ,int input_dim, int output_dim, int height_out, int width_out, int height_in, int width_in,  int kernal_size,int stride){

	int z = blockIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (col<width_out && row<height_out){
		float i = 0.0f;
		for (int d=0; d<input_dim; d++){
			for (int kh=0;kh<kernal_size;kh++){
				for (int kw=0; kw<kernal_size;kw++){
					i+=weights[(z*input_dim*kernal_size*kernal_size)+(d*kernal_size*kernal_size)+(kh*kernal_size)+kw] * input[(d*width_in*height_in)+(row*(stride)*width_in)+(col*(stride))+kw+(kh*width_in)];
				}
			}
		}

		output[(z * height_out * width_out) + (row * width_out) + col] = i + bias[z];
	}
}

Tensor Conv2d::forward(Tensor& input){
	input.to_device();
	input.add_padding(padding,0);
	const std::vector<int> input_shape = input.get_shape();
	int input_dim = input_shape[input_shape.size()-3];
	int height_in = input_shape[input_shape.size()-2];
	int width_in = input_shape[input_shape.size()-1];

	int height_out = 0;
	int width_out = 0;

	for(int f1=kernal-1; f1<height_in; f1+=stride){
		height_out++;
	}

	for(int f1=kernal-1; f1<width_in; f1+=stride){
		width_out++;
	}

	float* add_X = input.device_address();
	float* add_W = weights.device_address();
	float* add_B = bias.device_address();

	int total_size_output = num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

	dim3 blockDim(32,32);
	dim3 gridDim((width_out+31)/32,(height_out+31)/32,num_outputs);
	convolution<<<gridDim, blockDim>>>(add_X, add_W, add_B, add_output, input_dim, num_outputs, height_out, width_out, height_in, width_in,  kernal, stride);

	hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
	std::vector<int> output_shape = {num_outputs,height_out,width_out};
	Tensor t_output(output_shape);
	t_output.from_list(output);

	return t_output;
}

