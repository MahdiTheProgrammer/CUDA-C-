#include "hip/hip_runtime.h"
#include "layer.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void convolution(int batch_size, float* input,float*weights, float* bias, float* output ,int input_dim, int output_dim, int height_out, int width_out, int height_in, int width_in,  int kernal_size,int stride){

	int z = blockIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int which_kernal = z%output_dim;
	int which_batch = z/output_dim;
	if (col<width_out && row<height_out){
		float i = 0.0f;
		for (int d=0; d<input_dim; d++){
			for (int kh=0;kh<kernal_size;kh++){
				for (int kw=0; kw<kernal_size;kw++){
					i+=weights[(which_kernal*input_dim*kernal_size*kernal_size)+(d*kernal_size*kernal_size)+(kh*kernal_size)+kw] * input[(which_batch*width_in*height_in*input_dim)+(d*width_in*height_in)+(row*(stride)*width_in)+(col*(stride))+kw+(kh*width_in)];
				}
			}
		}

		output[(which_batch * output_dim * width_out * height_out)+(which_kernal * height_out * width_out) + (row * width_out) + col] = i + bias[which_kernal];
	}
}

Tensor Conv2d::forward(Tensor& input){
	input.to_device();
	input.add_padding(padding,0);
	const std::vector<int> input_shape = input.get_shape();

	int batch_size = input_shape[input_shape.size()-4];
	int input_dim = input_shape[input_shape.size()-3];
	int height_in = input_shape[input_shape.size()-2];
	int width_in = input_shape[input_shape.size()-1];

	int height_out = 0;
	int width_out = 0;

	for(int f1=kernal-1; f1<height_in; f1+=stride){
		height_out++;
	}

	for(int f1=kernal-1; f1<width_in; f1+=stride){
		width_out++;
	}

	float* add_X = input.device_address();
	float* add_W = weights.device_address();
	float* add_B = bias.device_address();

	int total_size_output = batch_size * num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

	dim3 blockDim(32,32);
	dim3 gridDim((width_out+31)/32,(height_out+31)/32,num_outputs*batch_size);
	convolution<<<gridDim, blockDim>>>(batch_size, add_X, add_W, add_B, add_output, input_dim, num_outputs, height_out, width_out, height_in, width_in,  kernal, stride);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
    		std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
	}
	hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
	std::vector<int> output_shape = {batch_size, num_outputs,height_out,width_out};
	Tensor t_output(output_shape);
	t_output.from_list(output);

	return t_output;
}

