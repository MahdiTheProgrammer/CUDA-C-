#include "hip/hip_runtime.h"
#include "layer.h"
#include <hip/hip_runtime.h>


__global__ void convolution2d(float* input,float*kernal, float* bias, int output, int x_out, int y_out){

	int z = gridIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
 

}

Tensor Conv2d::forward(Tensor& input){

	std::vecotr<int> input_shape = input.get_shape();

	int padded_height = input_shape[input_shape.size()-2]+ (padding * 2);
	int x_out = 0;

	int padded_width = input_shape[input_shape.size()-1] + (padding * 2);
	int y_out = 0;

	for(int f1=kernal; f1=<padded_height; f1+stride){
		x_out++;
	}

	for(int f1=kernal; f1=<padded_width; f1+stride){
		y_out++;
	}
	input.add_padding(padding,0);
	float* in = input.device_address();
	float* kernal = weights.device_address();
	float* b = bias.device_address();

	std::vector<int> output_shape = {num_output,x_out,y_out};
	dim3 blockDim(32,32);
	dim3 gridDim(output,(x_out+31)/32,(y_out+31)/32);
	convolution2d<<<gridDim, blockDim>>>(in, kernal, b, num_outputs, x_out, y_out);

	hipDeviceSynchronize();



	return output;
}
