#include "hip/hip_runtime.h"
#include "layer.h"
#include <hip/hip_runtime.h>


__global__ void convolution(float* input,float*weights, float* bias, float* output ,int input_dim, int output_dim, int height_out, int width_out, int height_in, int width_in,  int kernal_size,int stride){

	int z = gridIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int i = 0.0f;
	for (int d=0; d<input_dim; d++){
		for (int kh=0;kh<kernal_size;kh++){
			for (int kw=0; kw<kernal_size;kw++){
				i+=weights[(d*kernal_size*kernal_size)+(kh*kernal_size)+kw] * input[(d*width_in*height_in)+(row*(stride+1)*width_in)+(col*(stride+1))+kw+(kh*width_in)];
			}
		}
	}

	output[ (z * height_out * width_out) + (row * width_out) + col] = i + bias[z];

}

Tensor Conv2d::forward(Tensor& input){

	const std::vector<int> input_shape = input.get_shape();
	int input_dim = input_shape[input_shape.size()-3];
	int height_in = input_shape[input_shape.size()-2];
	int width_in = input_shape[input_shape.size()-1];

	int padded_height = input_shape[input_shape.size()-2]+ (padding * 2);
	int height_out = 0;

	int padded_width = input_shape[input_shape.size()-1] + (padding * 2);
	int width_out = 0;

	for(int f1=kernal; f1<=padded_height; f1+=stride){
		height_out++;
	}

	for(int f1=kernal; f1<=padded_width; f1+=stride){
		width_out++;
	}

	input.add_padding(padding,0);

	float* add_X = input.device_address();
	float* add_W = weights.device_address();
	float* add_B = bias.device_address();

	int total_size_output = num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

	std::vector<int> output_shape = {num_output,height_out,width_out};
	dim3 blockDim(32,32);
	dim3 gridDim(output,(height_out+31)/32,(width_out+31)/32);
	convolution<<<gridDim, blockDim>>>(add_X, add_W, add_B, add_output, input_dim, num_outputs, height_out, width_out, height_in, width_in,  kernal, stride);

	hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
	Tensor t_output(ouput_shape);
	output_shape.from_list(output);

	return t_output;
}

