#include "hip/hip_runtime.h"
#include "layer.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void maxpool(int batch_size, float* input, float* output, int height_out, int width_out, int height_in, int width_in, int kernel, int stride){

	int z = blockIdx.z;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row<height_out && col<width_out){
	float l = -INFINITY;
	float val;
		for(int h=0; h<kernel; h++){
			for(int w=0; w<kernel; w++){
				val = input[(z*height_in*width_in)+(row*stride*width_in)+(col*stride)+(h*width_in)+w];
				if(val>l){
					l=val;
				}
			}
		}
	output[(z * height_out * width_out)+ (row*width_out) + col] = l;
	}

}



Tensor MaxPool2d::forward(Tensor& input){
        input.to_device();

        const std::vector<int> input_shape = input.get_shape();
        int batch_size = input_shape[input_shape.size()-4];
        int input_dim = input_shape[input_shape.size()-3];
        int height_in = input_shape[input_shape.size()-2];
        int width_in = input_shape[input_shape.size()-1];

        int height_out = 0;
        int width_out = 0;

        for(int f1=kernel-1; f1<height_in; f1+=stride){
                height_out++;
        }

        for(int f1=kernel-1; f1<width_in; f1+=stride){
                width_out++;
        }

        float* add_X = input.device_address();

	int num_outputs = batch_size * input_dim;
	int total_size_output = batch_size * num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

        dim3 blockDim(32,32);
        dim3 gridDim((width_out+31)/32,(height_out+31)/32, num_outputs);
        maxpool<<<gridDim, blockDim>>>(batch_size, add_X, add_output, height_out, width_out, height_in, width_in,  kernel, stride);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
                std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
        std::vector<int> output_shape = {batch_size,input_dim,height_out,width_out};
        Tensor t_output(output_shape);
        t_output.from_list(output);

        return t_output;



}
