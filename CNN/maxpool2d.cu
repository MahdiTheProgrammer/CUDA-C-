#include "hip/hip_runtime.h"
#include <layer.h>
#include <hip/hip_runtime.h>

__global__ void maxpool2d(int batch_size, float* input, float* output,int kernel,int width_out, int height_out ){

	z = blockIdx.z;
	row = blockIdx.y * blockDim.y + threadIdx.y;
	col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row<height_out && col<width_out){
	float l;
		for(int h=0; h<kernel; h++){
			for(int w=0; w<kernel; w++){
				//
			}
		}
	output[(z * height_out * width_out)+ (row*width_out) + col] = l;
	}

}



Tensor MaxPool2D::forward(Tensor& input){
        input.to_device();

        const std::vector<int> input_shape = input.get_shape();
        int batch_size = input_shape[input_shape.size()-4];
        int input_dim = input_shape[input_shape.size()-3];
        int height_in = input_shape[input_shape.size()-2];
        int width_in = input_shape[input_shape.size()-1];

        int height_out = 0;
        int width_out = 0;

        for(int f1=kernel_size-1; f1<height_in; f1+=stride){
                height_out++;
        }

        for(int f1=kernal_size-1; f1<width_in; f1+=stride){
                width_out++;
        }

        float* add_X = input.device_address();

	num_outputs = batch_size * input_dim;
	int total_size_output = batch_size * num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

        dim3 blockDim(32,32);
        dim3 gridDim((width_out+31)/32,(height_out+31)/32, num_outputs);
        convolution<<<gridDim, blockDim>>>(batch_size, add_X, add_output, input_dim, num_outputs, height_out, width_out, height_in, width_in,  kernel, stride);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
                std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
        std::vector<int> output_shape = {batch_size, num_outputs,height_out,width_out};
        Tensor t_output(output_shape);
        t_output.from_list(output);

        return t_output;



}
