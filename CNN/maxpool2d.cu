#include "hip/hip_runtime.h"
#include <layer.h>

__global__ void maxpool2d(){


}



Tensor MaxPool2D::forward(Tensor& input){
        input.to_device();

        const std::vector<int> input_shape = input.get_shape();
        int batch_size = input_shape[input_shape.size()-4];
        int input_dim = input_shape[input_shape.size()-3];
        int height_in = input_shape[input_shape.size()-2];
        int width_in = input_shape[input_shape.size()-1];

        int height_out = 0;
        int width_out = 0;

        for(int f1=kernel_size-1; f1<height_in; f1+=stride){
                height_out++;
        }

        for(int f1=kernal_size-1; f1<width_in; f1+=stride){
                width_out++;
        }

        float* add_X = input.device_address();

	num_outputs = batch_size * input_dim;
	int total_size_output = batch_size * num_outputs * height_out * width_out;
        float* add_output;
        float* output = new float[total_size_output];
        hipMalloc((void**)&add_output,total_size_output * sizeof(float));

        dim3 blockDim(32,32);
        dim3 gridDim((width_out+31)/32,(height_out+31)/32, num_outputs);
        convolution<<<gridDim, blockDim>>>(batch_size, add_output, input_dim, num_outputs, height_out, width_out, height_in, width_in,  kernel, stride);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
                std::cerr << "CUDA kernel failed: " << hipGetErrorString(err) << std::endl;
        }
        hipDeviceSynchronize();

        hipMemcpy(output, add_output, total_size_output * sizeof(float), hipMemcpyDeviceToHost);
        std::vector<int> output_shape = {batch_size, num_outputs,height_out,width_out};
        Tensor t_output(output_shape);
        t_output.from_list(output);

        return t_output;



}
