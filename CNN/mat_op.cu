#include "hip/hip_runtime.h"
//Contains all CUDA kernels (e.g. matrixMultiply, ReLU)
#include <iostream>
#include <hip/hip_runtime.h>
#include "mat_op.h"
#include "utils.h"

__global__ void matrixmultiplication(float *t_A, float *t_B, float *c, int batch_size, int m, int n, int k){
	int batch_id = blockIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int i = (batch_id * m * k) + row * k + col;
	float e=0.0f;
	if (row < m && col < k){
		for(int f1=0; f1<n;f1++){
			e += t_A[ (batch_id * m  * n) + n*row +f1 ] * t_B[ ( batch_id * n  * k ) + (k*f1) + col];
//			e += t_A[(batch_id * shape_A[shape_A.size()-2)] * shape_A[shape_A.size()-1)]) + shape_A[shape_A.size()-1]*row +f1] * t_B[(batch_id * shape_B[shape_B.size()-2)] * shape_B[shape_B.size()-1)]) + (shape_B[shape_B.size()-1]*f1) + col];
		}
		c[i] = e;
	}
}



float* Tensor::matmul(const Tensor& t_A, const Tensor& t_B){
        std::vector<int> shape_A = t_A.get_shape();
        std::vector<int> shape_B = t_B.get_shape();
        int d=1;
        std::vector<int> shape = t_A.get_shape();
        for(int f1=0; f1<shape.size()-2;f1++){
                d*=shape_A[f1];
        }
        float* add_A = t_A.device_address();
        float* add_B = t_B.device_address();
        int total_size_C = d * shape_B[shape_B.size()-1] * shape_A[shape_A.size()-2];
        float *add_C;
        float *h_C = new float[total_size_C];
        hipMalloc((void**)&add_C,total_size_C * sizeof(float));
        dim3 blockDim(32,32);
        dim3 gridDim((shape_B[shape_B.size()-1]+31)/32,(shape_A[shape_A.size()-2]+31)/32 , d);
        matrixmultiplication<<<gridDim, blockDim>>>(add_A,add_B,add_C,d,shape_A[shape_A.size()-2],shape_B[shape_B.size()-2],shape_B[shape_B.size()-1]);
        hipDeviceSynchronize();
        hipMemcpy(h_C, add_C, total_size_C * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(add_C);
        return h_C;
}


