#include "hip/hip_runtime.h"
//Contains all CUDA kernels (e.g. matrixMultiply, ReLU)



__global__ void matmul(float *t_A, float *t_B, float *c, std::vecotr<int>& shape_A, std::vector<int> shape_B){
	int batch_id = blockIdx.z;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int i = (batch_id * shape_A[shape_A.size()-2)] * shape_B[shape_B.size()-1)]) + row * shape_B[shape_B.size()-1] + col;
	int e=0;
	if (row < shape_A[shape_A.size()-2] && col < shape_B[shape_B.size-1]){
		for(int f1=0; f1<shape_A[shape_A.size()-1];f1++){
			e += t_A[(batch_id * shape_A[shape_A.size()-2)] * shape_A[shape_A.size()-1)]) + shape_A[shape_A.size()-1]*row +f1] * t_B[(batch_id * shape_B[shape_B.size()-2)] * shape_B[shape_B.size()-1)]) + (shape_B[shape_B.size()-1]*f1) + col]
		}
		c[i] = e;
	}
}

