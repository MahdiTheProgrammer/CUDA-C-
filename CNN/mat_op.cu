#include "hip/hip_runtime.h"
//Contains all CUDA kernels (e.g. matrixMultiply, ReLU)



__global__ void matmul(float *t_A, float *t_B, float *c, std::vecotr<int>& s_A, std::vector<int> ){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int e=0;
	if (row < x && col < y){
		for(int f1=0; f1<m;f1++){
			e += a[m*row +f1] * b[col + f1*y];
		}
		c[y*row + col] = e; 
	}
}

//this only supports 2d tensros atm, but it will be soon working with any dim.
