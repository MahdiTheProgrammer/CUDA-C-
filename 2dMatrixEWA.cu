#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addmatrix(int *a, int *b, int *c, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i< N*N){
		c[i] = a[i] + b[i];
	}

}

int main(){
	const int N = 64;
	const int size = N * N * sizeof(int);

	int *h_a = new int[N*N];
	int *h_b = new int[N*N];
	int *h_c = new int[N*N];

	int *d_a, *d_b, *d_c;

	// I am just assiging some value to the matrices, doesn't really matter what.
	for(int f1 = 0; f1<64; f1++){
		for(int f2 = 0; f2<64; f2++){
			h_a[f1 * (N) + f2] = f1;
			h_b[f1 * (N) + f2] = f2;
		}
	}

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

	int ThreadperBlock = 1024;
	int BlockperGrid = N*N / 1024;

	addmatrix<<<BlockperGrid, ThreadperBlock>>>(d_a, d_b, d_c, N);

	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	bool success = true;
	for(int f1 = 0; f1<N; f1++){
		for(int f2 = 0; f2<N; f2++){
			if (h_c[f1 * (N) + f2] != f1+f2){
				printf("Failed");
				success = false;
				break;
			}
		}
	}

	if (success) {
                std::cout << "Vector addition successful!\n";
        }


	delete[] h_a;
	delete[] h_b;
	delete[] h_c;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
// Not Ready yet
