#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void matrixmultiplication(int *a, int *b, int *c, int x,int m, int y){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int e=0;
	if (row < x && col < y){
		for(int f1=0; f1<m;f1++){
			e += a[m*row +f1] * b[col + f1*y];
		}
		c[y*row + col] = e; 
	}
}

int main(){

	// Initializing two matrices with shape of x,m and m,y
	int x = 23000;
	int m = 30000;
	int y = 20000;

	int size_A = x*m*sizeof(int);
	int size_B = m*y*sizeof(int);
	int size_C = x*y*sizeof(int);

	int *h_A = new int[x*m];
	int *h_B = new int[m*y];
	int *h_C = new int[x*y];

	for (int f1=0; f1<x; f1++){
		for (int f2=0; f2<m;f2++){
			h_A[f1*m + f2] = 1;
			// h_A[f1*m + f2] = f1*m +f2+1;
		}
	}


	for (int f1=0; f1<m; f1++){
		for (int f2=0; f2<y;f2++){
			h_B[f1*y + f2] = f1*y + f2 +1;
		}
	}


	for (int f1=0; f1<x; f1++){
		for (int f2=0; f2<m; f2++){
			cout << h_A[f1*m + f2] << ",";
		}
		cout << endl;
	}
	cout << endl;

	for (int f1=0; f1<m; f1++){
		for (int f2=0; f2<y; f2++){
			cout << h_B[f1*y + f2] << ",";
		}
		cout << endl;
	}
	cout << endl;

	int *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, size_A);
	hipMalloc((void**)&d_B, size_B);
	hipMalloc((void**)&d_C, size_C);

	hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);

	dim3 blockDim(32,32);
	dim3 gridDim((y+31)/32,(x+31)/32);
	matrixmultiplication<<<gridDim,blockDim>>>(d_A,d_B,d_C,x,m,y);
	hipDeviceSynchronize();

	hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

	for (int f1=0; f1<x; f1++){
		for (int f2=0; f2<y; f2++){
			cout << h_C[f1*y + f2] << ",";
		}
		cout << endl;
	}
	
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
