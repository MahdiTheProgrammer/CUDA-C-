
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<N) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	const int N = 1 << 8;
	const int size = N * sizeof(int);

	int *h_a = new int[N];
	int *h_b = new int[N];
	int *h_c = new int[N];

	for (int i = 0; i<N; i++){
		h_a[i] = i;
		h_b[i] = i+1;
	}

	int *d_a, *d_b, *d_c;

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

	int ThreadsPerBlock = 512;
	int BlocksPerGrid = (N + ThreadsPerBlock -1) / ThreadsPerBlock;

	vectorAdd<<<BlocksPerGrid,ThreadsPerBlock>>>(d_a,d_b,d_c, N);

	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

	bool success = true;
    	for (int i = 0; i < N; i++) {
		if (h_c[i] != h_a[i] + h_b[i]) {
        	    	success = false;
           		std::cout << "Error at index " << i << ": " << h_c[i] << " != " << h_a[i] + h_b[i] << "\n";
            		break;
        	}
    	}
	if (success) {
		std::cout << "vector addition successful\n";
	}

	delete[] h_a;
	delete[] h_b;
	delete[] h_c;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
